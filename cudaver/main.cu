
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <png.h>
#include <stdlib.h>

__device__ double distance(double x, double y) {
    return abs((x * x + y * y) - 4);
}

__global__ void calc(int imgx, int imgy, char *cmem) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > imgx * imgy) {
        return;
    }
    int x = i % imgx;
    int y = i / imgx;

    double sx = -0.4;
    double ex = 0.1;
    double sy = 0.67;
    double ey = 1.17;

    double cx = (double)x / imgx * (ex - sx) + sx;
    double cy = (double)y / imgy * (ey - sy) + sy;

    double zx = 0;
    double zy = 0;

    double d = 1e6;

    for(int j = 0; j < 10000; j++) {
        double tx = zx;
        zx = zx * zx - zy * zy + cx;
        zy = 2 * tx * zy + cy;
        if (d > distance(zx, zy)) {
            d = distance(zx, zy);
        }
    }

    char c = 255 / (1 + d);
    if (distance(zx, zy) < 4) {
        c = 0;
    }
    cmem[3 * i + 0] = c;
    cmem[3 * i + 1] = c;
    cmem[3 * i + 2] = c;
}

int main(void) {
    int imgx = 4000;
    int imgy = 4000;

    char *cmem;

    hipMalloc(&cmem, sizeof(char) * imgx * imgy * 3);

    int bs = 32;

    calc<<<(imgx*imgy+bs-1)/bs, bs>>>(imgx, imgy, cmem);


    // open write file
    // https://daeudaeu.com/libpng/ のpngFileEncodeWriteを参考
    FILE *fo;
    int j;

    png_structp png;
    png_infop info;
    png_bytepp datap;
    png_byte type;

    fo = fopen("img.png", "wb");
    if (fo == NULL) {
        return 1;
    }

    png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    info = png_create_info_struct(png);

    type = PNG_COLOR_TYPE_RGB;

    png_init_io(png, fo);
    png_set_IHDR(png, info, imgx, imgy, 8, type, PNG_INTERLACE_NONE, PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);

    datap = (png_bytepp)png_malloc(png, sizeof(png_bytep) * imgy);

    png_set_rows(png, info, datap);

    for (j = 0; j < imgy; j++) {
        datap[j] = (png_bytep)png_malloc(png, 3 * imgx);

        // cmemはyが反転しているので上下反転する
        hipMemcpy(datap[j], cmem + 3 * (imgy - j - 1) * imgx, 3 * imgx, hipMemcpyDefault);
    }
    png_write_png(png, info, PNG_TRANSFORM_IDENTITY, NULL);

    hipFree(cmem);

    for (j = 0; j < imgy; j++) {
        png_free(png, datap[j]);
    }
    png_free(png, datap);

    png_destroy_write_struct(&png, &info);
    fclose(fo);

    return 0;
}